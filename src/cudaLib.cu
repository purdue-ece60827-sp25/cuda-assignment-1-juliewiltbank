#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

#define MAX_VAL 4294967295U

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int verifyGPUVector(float* a, float* b, float* c, float scale, int size) {
	int errorCount = 0;
	float relativeTolerance = 1e-6;
	float absoluteTolerance = 1e-7;
	for (int idx = 0; idx < size; ++idx) {
		float expected = scale * a[idx] + b[idx];
		float diff = abs(c[idx] - expected);
		if (diff > absoluteTolerance && diff > relativeTolerance * abs(expected)) {
			std::cout << abs(c[idx] - expected) << '\n';
			++errorCount;
			#ifndef DEBUG_PRINT_DISABLE
				std::cout << "Idx " << idx << " expected " << scale * a[idx] + b[idx] 
					<< " found " << c[idx] << " = " << a[idx] << " + " << b[idx] << "\n";
			#endif
		}
	}
	return errorCount;
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
		y[i] = scale * x[i] + y[i];
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	srand(time(0));
	float scale = static_cast<float>(rand() % MAX_VAL);
	float *x, *y, *result, *d_x, *d_y;
	x = (float*)malloc(vectorSize*sizeof(float));
	y = (float*)malloc(vectorSize*sizeof(float));
	result = (float*)malloc(vectorSize*sizeof(float));

	if (x == NULL || y == NULL || result == NULL) {
		printf("Unable to malloc memory ... Exiting!");
		return -1;
	}

	hipMalloc(&d_x, vectorSize*sizeof(float));
	hipMalloc(&d_y, vectorSize*sizeof(float));

	for (uint64_t i=0; i < vectorSize; i++) {
		x[i] = static_cast<float>(rand() % MAX_VAL);
		y[i] = static_cast<float>(rand() % MAX_VAL);
	}

	#ifndef DEBUG_PRINT_DISABLE 
		printf("\n Adding vectors : \n");
		printf(" scale = %3.4f\n", scale);
		printf(" x = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", x[i]);
		}
		printf(" ... }\n");
		printf(" y = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", y[i]);
		}
		printf(" ... }\n");
	#endif

	hipMemcpy(d_x, x, vectorSize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, vectorSize*sizeof(float), hipMemcpyHostToDevice);

	dim3 DimGrid(ceil(vectorSize/256.0), 1, 1);
	if (vectorSize%256) DimGrid.x++;
	dim3 DimBlock(256, 1, 1);

	saxpy_gpu<<<DimGrid,DimBlock>>>(d_x, d_y, scale, vectorSize);

	hipMemcpy(result, d_y, vectorSize*sizeof(float), hipMemcpyDeviceToHost);

	#ifndef DEBUG_PRINT_DISABLE 
		printf(" y = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", result[i]);
		}
		printf(" ... }\n");
	#endif

	int errorCount = verifyGPUVector(x, y, result, scale, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	// Insert code here
	hiprandState_t rng;
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= pSumSize) return;
	hiprand_init(clock64(), threadId, 0, &rng);

	uint64_t hitCount = 0;
	float x, y;

	for (uint64_t idx = 0; idx < sampleSize; ++idx) {
		x = hiprand_uniform(&rng);
		y = hiprand_uniform(&rng);
			
		if ( int(x * x + y * y) == 0 ) {
			++ hitCount;
		}
	}

	pSums[threadId] = hitCount;
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= reduceSize) return;

	uint64_t localSum = 0;
	for (uint64_t i = threadId; i < pSumSize; i += reduceSize) {
		localSum += pSums[i];
	}

	totals[threadId] = localSum;
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	uint64_t *totals, *d_pSums, *d_totals;
	totals = (uint64_t*)malloc(sampleSize*sizeof(uint64_t));

	if (totals == NULL) {
		printf("Unable to malloc memory ... Exiting!");
		return -1;
	}

	hipMalloc(&d_pSums, sampleSize*sizeof(uint64_t));
	hipMalloc(&d_totals, sampleSize*sizeof(uint64_t));

	dim3 DimGrid(ceil(sampleSize/256.0), 1, 1);
	if (sampleSize%256) DimGrid.x++;
	dim3 DimBlock(256, 1, 1);

	generatePoints<<<DimGrid,DimBlock>>>(d_pSums, generateThreadCount, sampleSize);

	reduceCounts<<<DimGrid,DimBlock>>>(d_pSums, d_totals, generateThreadCount, reduceThreadCount);

	hipMemcpy(totals, d_totals, sampleSize*sizeof(uint64_t), hipMemcpyDeviceToHost);

	uint64_t totalHitCount = 0;
    for (uint64_t i = 0; i < reduceThreadCount; ++i) {
        totalHitCount += totals[i];
    }

	hipFree(d_pSums);
	hipFree(d_totals);
	free(totals);

	approxPi = ((double)totalHitCount / sampleSize) / generateThreadCount ;
	approxPi = approxPi * 4.0f;
		
	return approxPi;
}
